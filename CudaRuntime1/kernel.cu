#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <fstream>
#include <chrono>
#include <cmath>
#include <tuple>
#include <string>

// Kernel CUDA do obliczeń
__global__ void calculateOnGPU(const float* input, float* output, int N, int R) {
    int outSize = N - 2 * R;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < outSize && j < outSize) {
        float sum = 0.0f;
        for (int x = -R; x <= R; ++x) {
            for (int y = -R; ++y <= R;) {
                sum += input[(i + R + x) * N + (j + R + y)];
            }
        }
        output[i * outSize + j] = sum;
    }
}

void checkCudaError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        std::cerr << msg << " Error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void loadDataFromFile(std::vector<float>& data, const std::string& filename) {
    std::ifstream inFile(filename, std::ios::binary);
    if (!inFile) {
        std::cerr << "Cannot open file for reading: " << filename << std::endl;
        exit(1);
    }
    inFile.read(reinterpret_cast<char*>(data.data()), data.size() * sizeof(float));
    inFile.close();
}

void loadParamsFromFile(int& N, int& R, const std::string& filename) {
    std::ifstream inFile(filename);
    if (!inFile) {
        std::cerr << "Cannot open file for reading: " << filename << std::endl;
        exit(1);
    }
    inFile >> N >> R;
    inFile.close();
}

bool compareResults(const std::vector<float>& cpuResults, const std::vector<float>& gpuResults, float tolerance = 1e-5) {
    if (cpuResults.size() != gpuResults.size()) {
        return false;
    }
    for (size_t i = 0; i < cpuResults.size(); ++i) {
        if (std::fabs(cpuResults[i] - gpuResults[i]) > tolerance) {
            std::cout << "Difference at index " << i << ": CPU = " << cpuResults[i] << ", GPU = " << gpuResults[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    int N, R;
    loadParamsFromFile(N, R, "params.txt");

    int inputSize = N * N;
    int outputSize = (N - 2 * R) * (N - 2 * R);
    int totalOps = outputSize * (2 * R + 1) * (2 * R + 1);

    std::vector<float> input(inputSize);
    std::vector<float> output(outputSize, 0.0f);
    std::vector<float> cpuOutput(outputSize, 0.0f);

    // Wczytanie danych wejściowych z pliku
    loadDataFromFile(input, "input_data.bin");

    // Wczytanie wyników CPU z pliku
    loadDataFromFile(cpuOutput, "cpu_output.bin");

    float* d_input, * d_output;
    checkCudaError(hipMalloc((void**)&d_input, inputSize * sizeof(float)), "Failed to allocate device input memory");
    checkCudaError(hipMalloc((void**)&d_output, outputSize * sizeof(float)), "Failed to allocate device output memory");

    checkCudaError(hipMemcpy(d_input, input.data(), inputSize * sizeof(float), hipMemcpyHostToDevice), "Failed to copy input data to device");

    std::vector<int> blockSizes = { 8, 16, 32 }; // Rozmiary bloków wątków

    // Zbiorcze wyniki
    std::vector<std::tuple<int, int, int, double, double, double>> results;

    for (int BS : blockSizes) {
        dim3 threadsPerBlock(BS, BS);
        dim3 numBlocks((N - 2 * R + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (N - 2 * R + threadsPerBlock.y - 1) / threadsPerBlock.y);

        auto start = std::chrono::high_resolution_clock::now();

        calculateOnGPU << <numBlocks, threadsPerBlock >> > (d_input, d_output, N, R);

        checkCudaError(hipGetLastError(), "Kernel launch failed");
        checkCudaError(hipDeviceSynchronize(), "Kernel synchronization failed");

        auto end = std::chrono::high_resolution_clock::now();

        checkCudaError(hipMemcpy(output.data(), d_output, outputSize * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy output data to host");

        std::chrono::duration<double> duration = end - start;
        double seconds = duration.count();
        double flops = totalOps / seconds;
        double cgma = static_cast<double>(totalOps) / (inputSize * sizeof(float) + outputSize * sizeof(float));

        // Zapis wyników do zbiorczych wyników
        results.push_back(std::make_tuple(N, R, BS, seconds, flops, cgma));

        // Wyświetlenie wyników
        /*
        std::cout << "Output Array (GPU, BS = " << BS << "):" << std::endl;
        for (int i = 0; i < outputSize; ++i) {
            if (i % (N - 2 * R) == 0) std::cout << std::endl;
            std::cout << output[i] << " ";
        }
        */
        std::cout << std::endl;

        std::cout << "Czas obliczeń: " << seconds << " seconds" << std::endl;
        std::cout << "Wydajność obliczeń: " << flops << " FLOP/s" << std::endl;
        std::cout << "Arithmetic Intensity (CGMA): " << cgma << " FLOP/byte" << std::endl;
        std::cout << "Rozmiar pamięci współdzielonej przez blok wątków: " << 0 << " bytes (not used)" << std::endl;

        // Porównanie wyników
        /*
        if (compareResults(cpuOutput, output)) {
            std::cout << "Wyniki obliczeń są poprawne!" << std::endl;
        }
        else {
            std::cout << "Wyniki obliczeń są niepoprawne!" << std::endl;
        }
        */
    }

    hipFree(d_input);
    hipFree(d_output);

    // Wyświetlenie zbiorczych wyników
    std::cout << "\nZbiorcze wyniki:" << std::endl;
    std::cout << "N\tR\tBS\tCzas(s)\tFLOP/s\tCGMA" << std::endl;
    for (const auto& result : results) {
        std::cout << std::get<0>(result) << "\t" << std::get<1>(result) << "\t" << std::get<2>(result) << "\t"
            << std::get<3>(result) << "\t" << std::get<4>(result) << "\t" << std::get<5>(result) << std::endl;
    }

    return 0;
}
